#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpu_Heat (float *h, float *g, float *r, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    bool fr = threadIdx.x == 0 && blockIdx.x == 0;
    bool fc = threadIdx.y == 0 && blockIdx.y == 0;

    /* if (i < N - 1 && j < N - 1 && !border_right && !border_top) { */
    float diff;
    int k = i * N + j;
    if (i < N - 1 && j < N - 1 && !fr && !fc) {
        //               left        top       right     bottom
        g[k] = 0.25 * (h[k - 1] + h[k - N] + h[k + 1] + h[k + N]);
        diff = g[k] - h[k];
        r[k] = diff * diff;
    }
    else
        r[k] = 0.0;
}

/* __global__ void gpu_residual (float *h, float *g, float *r, int N) { */
/*     int i = blockIdx.x * blockDim.x + threadIdx.x; */
/*     int j = blockIdx.y * blockDim.y + threadIdx.y; */

/*     bool fr = threadIdx.x == 0 && blockIdx.x == 0; // top line from the grid */
/*     bool fc = threadIdx.y == 0 && blockIdx.y == 0; // leftest line from the grid */

/*     float diff; */

/*     int k = i * N + j; */
/*     if (i < N - 1 && j < N - 1 && !fr && !fc) { */
/*         diff = g[k] - h[k]; */
/*         r[k] = diff * diff; */
/*     } */
/*     else */
/*         r[k] = 0.0; */
/*     /1* if(i == 1 && j == 1) printf("k %d, g %f, h %f, r %f, n %d \n",k, g[k], h[k], r[k], N); *1/ */
/* } */

__global__ void reduce0 (float *g_in, float *g_out/*, int np*/, unsigned int N) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = i < N ? g_in[i] : 0.0;
    __syncthreads();


    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) 
        g_out[blockIdx.x] = sdata[0];
    /* if (i == (0)) printf("reduce: k %d, residual reduced %f, g_in %f, n %d \n", 0, sdata[0], g_in[0], N); */
}

__global__ void reduce1 (float *g_in, float *g_out, unsigned int N) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = i < N ? g_in[i] : 0.0;
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        /* if (tid % (2 * s) == 0) { */
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) 
        g_out[blockIdx.x] = sdata[0];
}
