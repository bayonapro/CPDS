#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (float *h, float *g, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    bool fr = threadIdx.x == 0 && blockIdx.x == 0;
    bool fc = threadIdx.y == 0 && blockIdx.y == 0;

    if (i < N - 1 && j < N - 1 && !fr && !fc) {
        int k = i * N + j;
        //               left        top       right     bottom
        g[k] = 0.25 * (h[k - 1] + h[k - N] + h[k + 1] + h[k + N]);
    }
}

#if REDUCTION
__global__ void gpu_residual (float *h, float *g, float *r, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    bool fr = threadIdx.x == 0 && blockIdx.x == 0;
    bool fc = threadIdx.y == 0 && blockIdx.y == 0;

    float diff;

    if (i < N - 1 && j < N - 1 && !fr && !fc) {
        int k = i * N + j;
        diff = g[k] - h[h];
        r[k] = diff * diff;
    }
}
#endif

#if REDUCTION == 1
__global__ void reduce0 (int *g_in, int *g_out) {
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = i < N ? g_in[i] : 0.0;
    __syncthreads();

    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            sdata[tid] += sdata[tid + s]
        }
        __syncthreads();
    }

    if (tid == 0) g_out[blockIdx.x] = sdata[0];
}
#endif
